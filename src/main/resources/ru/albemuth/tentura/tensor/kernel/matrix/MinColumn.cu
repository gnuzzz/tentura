#define TILE_DIM 1024


#include <hip/hip_runtime.h>
#include <limits>

template<typename T>
__device__ void minColumn(const T* matrix, T* result, const int numRows, const int numColumns) {

  __shared__ T threadMin[TILE_DIM];

  int index = threadIdx.x;
  int rowStride = blockDim.x;
  int partLength = (numColumns + TILE_DIM - 1) / TILE_DIM;
  int limit = numColumns < TILE_DIM ? numColumns : TILE_DIM;

  for (int row = blockIdx.x; row < numRows; row += rowStride) {

    T min = std::numeric_limits<T>::max();
    for (int i = 0; i < partLength; i++) {
      int columnIndex = i * TILE_DIM + index;
      if (columnIndex < numColumns) {
        T value = matrix[row * numColumns + columnIndex];
        if (value < min) {
          min = value;
        }
      }
    }
    threadMin[index] = min;

    for (int d = 1; d < limit; d <<= 1) {
      __syncthreads();
      if (index % (d << 1) == 0) {
        int valueIndex = index + d;
        if (valueIndex < limit) {
          T value = threadMin[valueIndex];
          if (value < min) {
            min = value;
            threadMin[index] = min;
          }
        }
      }
    }

    if (index == 0) {
      result[row] = min;
    }
  }
}